/* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include <cstddef>
#include <cinttypes>
#include <hip/device_functions.h>

static inline __device__ float
clamp(const float v, const float low, const float high) {
	return v < low ? low : v > high ? high : v;
}

static inline __device__ float
rgb2y(const uint8_t r, const uint8_t g, const uint8_t b) {
	return 0.299f*(float)r + 0.587f*(float)g + 0.114f*(float)b;
}
static inline __device__ float
rgb2u(const uint8_t r, const uint8_t g, const uint8_t b) {
	const float y = rgb2y(r,g,b);
	return clamp(-(((-(float)b + y) / 1.732446f) - 128.f), 0.f, 255.f);
}
static inline __device__ float
rgb2v(const uint8_t r, const uint8_t g, const uint8_t b) {
	const float y = rgb2y(r,g,b);
	const float u = rgb2u(r,g,b);
	return clamp((y - (0.337633f*(u-128.f)) - (float)g) / 0.698001f + 128.f,
	             0.f, 255.f);
}

/* Converts from RGB data to NV12.  NV12's "U" and "V" channels are interleaved
 * and subsampled 2x2.  Note the RGB data are not pitched. */
extern "C" __global__ void
rgb2yuv(const uint8_t* const __restrict rgb,
        const size_t width, const size_t height, const size_t c/*omponents*/,
        uint8_t* const __restrict yuv, unsigned pitch) {
	const size_t x = blockIdx.x*blockDim.x + threadIdx.x;
	const size_t y = blockIdx.y*blockDim.y + threadIdx.y;
	const size_t i = y*pitch + x;
	const size_t j = y*width + x;
	if(x >= width || y >= height || i >= pitch*height || j >= width*height) {
		return;
	}
	assert(pitch >= width);
	assert(i < pitch*height);
	assert(j < width*height);
	assert(width <= 4096);
	assert(height <= 4096);
	assert(c == 3 || c == 4);
	assert(pitch <= 4096);
	uint8_t* __restrict Y = yuv;
	Y[i] = (uint8_t)clamp(rgb2y(rgb[j*c+0], rgb[j*c+1], rgb[j*c+2]), 0, 255);
	/* U+V are downsampled 2x per dimension.  So kill off 3 of every 4 threads
	 * that reach here; only one will do the writes into U and V. */
	/* thought: use x0 to write into U and x1 to write into V, to spread load? */
	if(x&1 == 1 || y&1 == 1) {
		return;
	}
	uint8_t* __restrict uv = yuv + pitch*height;
	const size_t uvidx = y/2*(pitch/2) + x/2;
	const size_t idx[4] = {
		min((y+0)*width + x+0, width*height),
		min((y+0)*width + x+1, width*height),
		min((y+1)*width + x+0, width*height),
		min((y+1)*width + x+1, width*height),
	};
	const float u[4] = {
		rgb2u(rgb[idx[0]*c+0], rgb[idx[0]*c+1], rgb[idx[0]*c+2]),
		rgb2u(rgb[idx[1]*c+0], rgb[idx[1]*c+1], rgb[idx[1]*c+2]),
		rgb2u(rgb[idx[2]*c+0], rgb[idx[2]*c+1], rgb[idx[2]*c+2]),
		rgb2u(rgb[idx[3]*c+0], rgb[idx[3]*c+1], rgb[idx[3]*c+2])
	};
	const float v[4] = {
		rgb2v(rgb[idx[0]*c+0], rgb[idx[0]*c+1], rgb[idx[0]*c+2]),
		rgb2v(rgb[idx[1]*c+0], rgb[idx[1]*c+1], rgb[idx[1]*c+2]),
		rgb2v(rgb[idx[2]*c+0], rgb[idx[2]*c+1], rgb[idx[2]*c+2]),
		rgb2v(rgb[idx[3]*c+0], rgb[idx[3]*c+1], rgb[idx[3]*c+2])
	};
	uv[uvidx*2+0] = (uint8_t)clamp((u[0] + u[1] + u[2] + u[3]) / 4.0, 0, 255);
	uv[uvidx*2+1] = (uint8_t)clamp((v[0] + v[1] + v[2] + v[3]) / 4.0, 0, 255);
}

static inline __device__ float
yuv2r(const uint8_t y, const uint8_t u, const uint8_t v) {
	(void)u;
	return (y-16)*1.164f + (1.596f * (v-128));
}
static inline __device__ float
yuv2g(const uint8_t y, const uint8_t u, const uint8_t v) {
	return (y-16)*1.164f + (u-128)*-0.392f + (v-128)*-0.813f;
}
static inline __device__ float
yuv2b(const uint8_t y, const uint8_t u, const uint8_t v) {
	(void)v;
	return (y-16)*1.164f + (u-128)*2.017f;
}

/* Convert back from NV12 to RGB.   Note the RGB buffer is not pitched. */
extern "C" __global__ void
yuv2rgb(const uint8_t* const __restrict yuv,
        const size_t width, const size_t height, unsigned pitch,
        uint8_t* const __restrict rgb) {
	const size_t x = blockIdx.x*blockDim.x + threadIdx.x;
	const size_t y = blockIdx.y*blockDim.y + threadIdx.y;
	const size_t i = y*pitch + x;
	const size_t j = y*width + x;
	if(x >= width || y >= height || i >= pitch*height || j >= width*height) {
		return;
	}
	assert(i < pitch*height);
	assert(j < width*height);
	assert(width <= 4096);
	assert(height <= 4096);
	assert(pitch <= 4096);
	const uint8_t* __restrict Y = yuv;
	const uint8_t* __restrict uv = yuv + pitch*height;
	const size_t idx[4] = {
		min((y+0)/2*pitch/2 + (x+0)/2, height*pitch),
		min((y+0)/2*pitch/2 + (x+1)/2, height*pitch),
		min((y+1)/2*pitch/2 + (x+0)/2, height*pitch),
		min((y+1)/2*pitch/2 + (x+1)/2, height*pitch),
	};
	const uint8_t u[4] = {
		uv[idx[0]*2+0], uv[idx[1]*2+0], uv[idx[2]*2+0], uv[idx[3]*2+0]
	};
	const uint8_t v[4] = {
		uv[idx[0]*2+1], uv[idx[1]*2+1], uv[idx[2]*2+1], uv[idx[3]*2+1]
	};
	rgb[j*3+0] = clamp(
		(yuv2r(Y[i], u[0], v[0]) + yuv2r(Y[i], u[1], v[1]) +
		 yuv2r(Y[i], u[2], v[2]) + yuv2r(Y[i], u[3], v[3])) / 4.0, 0, 255
	);
	rgb[j*3+1] = clamp(
		(yuv2g(Y[i], u[0], v[0]) + yuv2g(Y[i], u[1], v[1]) +
		 yuv2g(Y[i], u[2], v[2]) + yuv2g(Y[i], u[3], v[3])) / 4.0, 0, 255
	);
	rgb[j*3+2] = clamp(
		(yuv2b(Y[i], u[0], v[0]) + yuv2b(Y[i], u[1], v[1]) +
		 yuv2b(Y[i], u[2], v[2]) + yuv2b(Y[i], u[3], v[3])) / 4.0, 0, 255
	);
}
